#include "hip/hip_runtime.h"
#include <cuda/helpers.h>
#include <hip/hip_runtime.h>
#include <optix.h>
#include <sutil/vec_math.h>
#include <cuda/std/complex>

#include "complex_vector.cu"
#include "rcs_params.h"

extern "C" {
	__constant__ Params params;

}



static __forceinline__ __device__ void trace(
	OptixTraversableHandle handle, float3 ray_origin, float3 ray_direction,
	unsigned int relCount, float tpath, float3 pol) {
	float tmin = 1e-5f;
	float tmax = 1e30f;
	unsigned path_uint = __float_as_uint(tpath);
	unsigned pol1 = __float_as_uint(pol.x);
	unsigned pol2 = __float_as_uint(pol.y);
	unsigned pol3 = __float_as_uint(pol.z);
	optixTrace(handle, ray_origin, ray_direction, tmin, tmax, 0.0f, OptixVisibilityMask(1),
		OPTIX_RAY_FLAG_DISABLE_ANYHIT, 0, 0, 0, relCount, path_uint, pol1, pol2, pol3);
}



extern "C" __global__ void __raygen__rg() {
	const uint3 idx = optixGetLaunchIndex();
	const uint3 dim = optixGetLaunchDimensions();
	int ray_id = idx.x + dim.x * idx.y;

	float3 origin;
	float3 direction;
	int idR = idx.x;
	int idU = idx.y;

	origin = params.rayPosBegin + params.rayPosStepU * idU + params.rayPosStepR * idR;
	direction = params.rayDir;

	trace(params.handle, origin, direction, 0u, 0.0f, params.polarization);
}



extern "C" __global__ void __miss__ms() {

	float3 ray_direction = optixGetWorldRayDirection();
	float3 ray_ori = optixGetWorldRayOrigin();

	const uint3 idx = optixGetLaunchIndex();
	const uint3 dim = optixGetLaunchDimensions();
	int ray_id = idx.x + dim.x * idx.y;

	float wave_num = params.wave_num;

	float phi = params.observer_angle.x;
	float the = params.observer_angle.y;

	float cp = cosf(phi);
	float sp = sinf(phi);
	float ct = cosf(the);
	float st = sinf(the);

	float3 dirX = make_float3(1.0, 0.0, 0.0);
	float3 dirY = make_float3(0.0, 1.0, 0.0);
	float3 dirZ = make_float3(0.0, 0.0, 1.0);
	float3 dirP = make_float3(-sp, cp, 0.0);
	float3 dirT = make_float3(cp * ct, sp * ct, -st);

	float3 vecK = wave_num * ((dirX * cp + dirY * sp) * st + dirZ * ct);

	using cuda::std::complex;
	complex<float> AU = 0;

	complex<float> AR = 0;
	complex<float> i = complex<float>(0.0f, 1.0f);
	float t_value = params.t_value;
	unsigned int refCount = optixGetPayload_0();
	if (refCount > 0) {

		float tpath = __uint_as_float(optixGetPayload_1());
		float kr = wave_num * tpath;

		float3 pol;
		pol.x = __uint_as_float(optixGetPayload_2());
		pol.y = __uint_as_float(optixGetPayload_3());
		pol.z = __uint_as_float(optixGetPayload_4());


		float relectance = params.reflectance;
		float reflectionCoef = powf(relectance, refCount);

		complexFloat3 apE = exp(i * kr) * pol * reflectionCoef;

		complexFloat3 apH = -cross(apE, ray_direction);

		complex<float> BU =
			dot(-(cross(apE, -dirP) + cross(apH, dirT)), ray_direction);

		complex<float> BR =
			dot(-(cross(apE, dirT) + cross(apH, dirP)), ray_direction);

		complex<float> e = exp(-i * dot(vecK, ray_ori));

		complex<float> factor = complex<float>(0.0, t_value) * e;


		AU = BU * factor;

		AR = BR * factor;
		/*if (pldptr->ray_id % 10000==0) {
			printf("factor: %f %f\n", factor.real(), factor.imag());
			printf("waveNum: %f\n", waveNum);
		}*/
	}
	params.result[4 * ray_id] = AU.real();
	params.result[4 * ray_id + 1] = AU.imag();
	params.result[4 * ray_id + 2] = AR.real();
	params.result[4 * ray_id + 3] = AR.imag();
}



extern "C" __global__ void __closesthit__triangle() {
	unsigned int tri_id = optixGetPrimitiveIndex();

	float3 ray_dir = optixGetWorldRayDirection();
	float3 ray_ori = optixGetWorldRayOrigin();

	float ray_tmax = optixGetRayTmax();

	float3 out_normal = params.out_normals[tri_id];

	float3 hit_point = ray_ori + ray_tmax * ray_dir;
	float3 reflect_dir = reflect(ray_dir, out_normal);

	float tpath = __uint_as_float(optixGetPayload_1());

	float3 pol;
	pol.x = __uint_as_float(optixGetPayload_2());
	pol.y = __uint_as_float(optixGetPayload_3());
	pol.z = __uint_as_float(optixGetPayload_4());

	float3 dirCrossNormal = cross(ray_dir, out_normal);

	float3 polU = normalize(dirCrossNormal);
	float3 polR = normalize(cross(ray_dir, polU));

	float3 refDir = reflect_dir;

	float3 refPolU = -polU;
	float3 refPolR = cross(refDir, refPolU);

	float polCompU = dot(pol, polU);
	float polCompR = dot(pol, polR);

	float total_path_length = ray_tmax + tpath;

	pol = -polCompR * refPolR + polCompU * refPolU;

	unsigned int refCount = optixGetPayload_0() + 1u;


	trace(params.handle, hit_point, reflect_dir, refCount, total_path_length, pol);
}