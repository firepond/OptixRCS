#include "hip/hip_runtime.h"
//#include <cuda/helpers.h>
#include <hip/hip_runtime.h>
#include "reduceKernels.h"
#include "rcs_params.h"
#include <sutil/vec_math.h>
#include <cmath>

__constant__ Result zero;


__global__ void reduceKernel(Result* g_idata, Result* g_odata, int size) {

	int temp = (blockIdx.x + 1) * blockDim.x;

	int curBlockSize = (temp <= size) * blockDim.x + (temp > size) * (size % blockDim.x);
	extern __shared__ Result sdata[512];

	// each thread loads one element from global to shared mem
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < curBlockSize) {
		sdata[tid] = g_idata[i];
	}
	else {
		sdata[tid] = zero;
	}

	__syncthreads();
	// do reduction in shared mem

	for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
		if (tid < s) {
			sdata[tid].ar_img += sdata[tid + s].ar_img;
			sdata[tid].ar_real += sdata[tid + s].ar_real;
			sdata[tid].au_img += sdata[tid + s].au_img;
			sdata[tid].au_real += sdata[tid + s].au_real;
	
		}
		__syncthreads();
	}

	// write result for this block to global mem
	if (tid == 0) {
		g_odata[blockIdx.x] = sdata[0];
	}

}


Result reduce(Result* g_idata, int size)
{
	int blockDim = 512;

	int reduceCount = ceil(log2(size) / log2(blockDim));

	Result* out_device;
	Result* to_reduce_device = g_idata;
	int block_count = ceil((double)size / blockDim);
	hipMalloc((void**)&out_device, sizeof(Result) * block_count);
	Result* out_device_holder = out_device;
	hipDeviceSynchronize();

	while (size > 1) {
		if (size <= blockDim) {
			reduceKernel <<< 1, blockDim >>> (to_reduce_device, out_device, size);
			break;
		}
		else {
			block_count = ceil((double)size / blockDim);
			reduceKernel <<< block_count, blockDim >>> (to_reduce_device, out_device, size);
		}
	
		hipDeviceSynchronize();

		// swap to_reduce_device and out_device
		Result* temp;
		temp = to_reduce_device;
		to_reduce_device = out_device;
		out_device = temp;

		size = block_count;
	}

	Result result_out;
	hipMemcpy(&result_out, out_device, sizeof(Result),
		hipMemcpyDeviceToHost);
	hipFree(reinterpret_cast<void*>(out_device_holder));
	hipDeviceSynchronize();
	return result_out;
}

