#include "hip/hip_runtime.h"
//
// Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions
// are met:
//  * Redistributions of source code must retain the above copyright
//    notice, this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright
//    notice, this list of conditions and the following disclaimer in the
//    documentation and/or other materials provided with the distribution.
//  * Neither the name of NVIDIA CORPORATION nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
// OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//

#include <optix.h>
#include <hip/hip_vector_types.h>

#include "helpers.h"
#include "whitted.h"

extern "C" {
__constant__ whitted::LaunchParams params;
}

static __device__ __inline__ whitted::PayloadRadiance getPayloadRadiance()
{
    whitted::PayloadRadiance prd;
    prd.result.x   = __uint_as_float( optixGetPayload_0() );
    prd.result.y   = __uint_as_float( optixGetPayload_1() );
    prd.result.z   = __uint_as_float( optixGetPayload_2() );
    prd.importance = __uint_as_float( optixGetPayload_3() );
    prd.depth      = optixGetPayload_4();
    return prd;
}

static __device__ __inline__ void setPayloadRadiance( const whitted::PayloadRadiance& prd )
{
    optixSetPayload_0( __float_as_uint( prd.result.x ) );
    optixSetPayload_1( __float_as_uint( prd.result.y ) );
    optixSetPayload_2( __float_as_uint( prd.result.z ) );
    optixSetPayload_3( __float_as_uint( prd.importance ) );
    optixSetPayload_4( prd.depth );
}

static __device__ __inline__ whitted::PayloadOcclusion getPayloadOcclusion()
{
    whitted::PayloadOcclusion prd;
    prd.result.x = __uint_as_float( optixGetPayload_0() );
    prd.result.y = __uint_as_float( optixGetPayload_1() );
    prd.result.z = __uint_as_float( optixGetPayload_2() );
    return prd;
}

static __device__ __inline__ void setPayloadOcclusion( const whitted::PayloadOcclusion& prd )
{
    optixSetPayload_0( __float_as_uint( prd.result.x ) );
    optixSetPayload_1( __float_as_uint( prd.result.y ) );
    optixSetPayload_2( __float_as_uint( prd.result.z ) );
}

static __device__ __inline__ float3 traceRadianceRay( float3 origin, float3 direction, int depth, float importance )
{
    whitted::PayloadRadiance prd;
    prd.depth      = depth;
    prd.importance = importance;

    optixTrace( params.handle, origin, direction, params.scene_epsilon, 1e16f, 0.0f, OptixVisibilityMask( 1 ), OPTIX_RAY_FLAG_NONE,
                whitted::RAY_TYPE_RADIANCE, whitted::RAY_TYPE_COUNT, whitted::RAY_TYPE_RADIANCE, float3_as_args( prd.result ),
                /* Can't use __float_as_uint() because it returns rvalue but payload requires a lvalue */
                reinterpret_cast<unsigned int&>( prd.importance ), reinterpret_cast<unsigned int&>( prd.depth ) );

    return prd.result;
}

static __device__ void phongShadowed()
{
    // this material is opaque, so it fully attenuates all shadow rays
    whitted::PayloadOcclusion prd;
    prd.result = make_float3( 0.f );
    setPayloadOcclusion( prd );
}

static __device__ void phongShade( float3 p_Kd, float3 p_Ka, float3 p_Ks, float3 p_Kr, float p_phong_exp, float3 p_normal )
{
    const float3 ray_orig = optixGetWorldRayOrigin();
    const float3 ray_dir  = optixGetWorldRayDirection();
    const float  ray_t    = optixGetRayTmax();

    whitted::PayloadRadiance prd = getPayloadRadiance();

    float3 hit_point = ray_orig + ray_t * ray_dir;

    // ambient contribution
    Light::Ambient ambient_light = params.lights[0].ambient;
    float3         result        = p_Ka * ambient_light.color;

    // compute direct lighting
    Light::Point point_light = params.lights[1].point;
    float        Ldist       = length( point_light.position - hit_point );
    float3       L           = normalize( point_light.position - hit_point );
    float        nDl         = dot( p_normal, L );

    // cast shadow ray
    float3 light_attenuation = make_float3( static_cast<float>( nDl > 0.0f ) );
    if( nDl > 0.0f )
    {
        whitted::PayloadOcclusion shadow_prd;
        shadow_prd.result = make_float3( 1.0f );

        optixTrace( params.handle, hit_point, L, 0.01f, Ldist, 0.0f, OptixVisibilityMask( 1 ), OPTIX_RAY_FLAG_NONE,
                    whitted::RAY_TYPE_OCCLUSION, whitted::RAY_TYPE_COUNT, whitted::RAY_TYPE_OCCLUSION,
                    float3_as_args( shadow_prd.result ) );

        light_attenuation = shadow_prd.result;
    }

    // If not completely shadowed, light the hit point
    if( fmaxf( light_attenuation ) > 0.0f )
    {
        float3 Lc = point_light.color * light_attenuation;

        result += p_Kd * nDl * Lc;

        float3 H   = normalize( L - ray_dir );
        float  nDh = dot( p_normal, H );
        if( nDh > 0 )
        {
            float power = pow( nDh, p_phong_exp );
            result += p_Ks * power * Lc;
        }
    }

    if( fmaxf( p_Kr ) > 0 )
    {

        // ray tree attenuation
        float new_importance = prd.importance * luminance( p_Kr );
        int   new_depth      = prd.depth + 1;

        // reflection ray
        // compare new_depth to max_depth - 1 to leave room for a potential shadow ray trace
        if( new_importance >= 0.01f && new_depth <= params.max_depth - 1 )
        {
            float3 R = reflect( ray_dir, p_normal );

            result += p_Kr * traceRadianceRay( hit_point, R, new_depth, new_importance );
        }
    }

    // pass the color back
    prd.result = result;
    setPayloadRadiance( prd );
}

extern "C" __global__ void __closesthit__checker_radiance()
{
    const whitted::HitGroupData*      sbt_data = (whitted::HitGroupData*)optixGetSbtDataPointer();
    const MaterialData::CheckerPhong& checker  = sbt_data->material_data.checker;

    float3 Kd, Ka, Ks, Kr;
    float  phong_exp;

    float2 texcoord = make_float2( __uint_as_float( optixGetAttribute_3() ), __uint_as_float( optixGetAttribute_4() ) );
    float2 t        = texcoord * checker.inv_checker_size;
    t.x             = floorf( t.x );
    t.y             = floorf( t.y );

    int which_check = ( static_cast<int>( t.x ) + static_cast<int>( t.y ) ) & 1;

    if( which_check )
    {
        Kd        = checker.Kd1;
        Ka        = checker.Ka1;
        Ks        = checker.Ks1;
        Kr        = checker.Kr1;
        phong_exp = checker.phong_exp1;
    }
    else
    {
        Kd        = checker.Kd2;
        Ka        = checker.Ka2;
        Ks        = checker.Ks2;
        Kr        = checker.Kr2;
        phong_exp = checker.phong_exp2;
    }

    float3 object_normal = make_float3( __uint_as_float( optixGetAttribute_0() ), __uint_as_float( optixGetAttribute_1() ),
                                        __uint_as_float( optixGetAttribute_2() ) );
    float3 world_normal = normalize( optixTransformNormalFromObjectToWorldSpace( object_normal ) );
    float3 ffnormal     = faceforward( world_normal, -optixGetWorldRayDirection(), world_normal );
    phongShade( Kd, Ka, Ks, Kr, phong_exp, ffnormal );
}

extern "C" __global__ void __closesthit__metal_radiance()
{
    const whitted::HitGroupData* sbt_data = (whitted::HitGroupData*)optixGetSbtDataPointer();
    const MaterialData::Phong&   phong    = sbt_data->material_data.metal;

    float3 object_normal = make_float3( __uint_as_float( optixGetAttribute_0() ), __uint_as_float( optixGetAttribute_1() ),
                                        __uint_as_float( optixGetAttribute_2() ) );

    float3 world_normal = normalize( optixTransformNormalFromObjectToWorldSpace( object_normal ) );
    float3 ffnormal     = faceforward( world_normal, -optixGetWorldRayDirection(), world_normal );
    phongShade( phong.Kd, phong.Ka, phong.Ks, phong.Kr, phong.phong_exp, ffnormal );
}

extern "C" __global__ void __closesthit__full_occlusion()
{
    phongShadowed();
}

extern "C" __global__ void __closesthit__glass_radiance()
{
    const whitted::HitGroupData* sbt_data = (whitted::HitGroupData*)optixGetSbtDataPointer();
    const MaterialData::Glass&   glass    = sbt_data->material_data.glass;

    whitted::PayloadRadiance prd_radiance = getPayloadRadiance();

    float3 object_normal = make_float3( __uint_as_float( optixGetAttribute_0() ), __uint_as_float( optixGetAttribute_1() ),
                                        __uint_as_float( optixGetAttribute_2() ) );
    object_normal        = normalize( object_normal );

    // intersection vectors
    const float3 n        = normalize( optixTransformNormalFromObjectToWorldSpace( object_normal ) );  // normal
    const float3 ray_orig = optixGetWorldRayOrigin();
    const float3 ray_dir  = optixGetWorldRayDirection();  // incident direction
    const float  ray_t    = optixGetRayTmax();
    float3       t;  // transmission direction
    float3       r;  // reflection direction

    float3                             hit_point       = ray_orig + ray_t * ray_dir;
    GeometryData::SphereShell::HitType hit_type        = (GeometryData::SphereShell::HitType)optixGetHitKind();
    float3                             front_hit_point = hit_point, back_hit_point = hit_point;

    if( hit_type & GeometryData::SphereShell::HIT_OUTSIDE_FROM_OUTSIDE || hit_type & GeometryData::SphereShell::HIT_INSIDE_FROM_INSIDE )
    {
        front_hit_point += params.scene_epsilon * object_normal;
        back_hit_point -= params.scene_epsilon * object_normal;
    }
    else
    {
        front_hit_point -= params.scene_epsilon * object_normal;
        back_hit_point += params.scene_epsilon * object_normal;
    }

    const float3 fhp = optixTransformPointFromObjectToWorldSpace( front_hit_point );
    const float3 bhp = optixTransformPointFromObjectToWorldSpace( back_hit_point );

    float  reflection = 1.0f;
    float3 result     = make_float3( 0.0f );

    const int depth = prd_radiance.depth;

    float3 beer_attenuation;
    if( dot( n, ray_dir ) > 0 )
    {
        // Beer's law attenuation
        beer_attenuation = exp( glass.extinction_constant * ray_t );
    }
    else
    {
        beer_attenuation = make_float3( 1 );
    }

    // refraction
    // compare depth to max_depth - 1 to leave room for a potential shadow ray trace
    if( depth < min( glass.refraction_maxdepth, params.max_depth - 1 ) )
    {
        if( refract( t, ray_dir, n, glass.refraction_index ) )
        {
            // check for external or internal reflection
            float cos_theta = dot( ray_dir, n );
            if( cos_theta < 0.0f )
                cos_theta = -cos_theta;
            else
                cos_theta = dot( t, n );

            reflection = fresnel_schlick( cos_theta, glass.fresnel_exponent, glass.fresnel_minimum, glass.fresnel_maximum );

            float importance =
                prd_radiance.importance * ( 1.0f - reflection ) * luminance( glass.refraction_color * beer_attenuation );
            float3 color = glass.cutoff_color;
            if( importance > glass.importance_cutoff )
            {
                color = traceRadianceRay( bhp, t, depth + 1, importance );
            }
            result += ( 1.0f - reflection ) * glass.refraction_color * color;
        }
        // else TIR
    }  // else reflection==1 so refraction has 0 weight

    // reflection
    // compare depth to max_depth - 1 to leave room for a potential shadow ray trace
    float3 color = glass.cutoff_color;
    if( depth < min( glass.reflection_maxdepth, params.max_depth - 1 ) )
    {
        r = reflect( ray_dir, n );

        float importance = prd_radiance.importance * reflection * luminance( glass.reflection_color * beer_attenuation );
        if( importance > glass.importance_cutoff )
        {
            color = traceRadianceRay( fhp, r, depth + 1, importance );
        }
    }
    result += reflection * glass.reflection_color * color;

    result = result * beer_attenuation;

    prd_radiance.result = result;
    setPayloadRadiance( prd_radiance );
}

extern "C" __global__ void __anyhit__glass_occlusion()
{
    const whitted::HitGroupData* sbt_data = (whitted::HitGroupData*)optixGetSbtDataPointer();
    const MaterialData::Glass&   glass    = sbt_data->material_data.glass;

    float3 object_normal = make_float3( __uint_as_float( optixGetAttribute_0() ), __uint_as_float( optixGetAttribute_1() ),
                                        __uint_as_float( optixGetAttribute_2() ) );

    whitted::PayloadOcclusion shadow_prd = getPayloadOcclusion();

    float3 world_normal = normalize( optixTransformNormalFromObjectToWorldSpace( object_normal ) );
    float  nDi          = fabs( dot( world_normal, optixGetWorldRayDirection() ) );

    shadow_prd.result *= 1 - fresnel_schlick( nDi, 5, 1 - glass.shadow_attenuation, make_float3( 1 ) );
    setPayloadOcclusion( shadow_prd );

    // Test the attenuation of the light from the glass shell
    if( luminance( shadow_prd.result ) < glass.importance_cutoff )
        // The attenuation is so high, > 99% blocked, that we can consider testing to be done.
        optixTerminateRay();
    else
        // There is still some light coming through the glass shell that we should test other occluders.
        // We "ignore" the intersection with the glass shell, meaning that shadow testing will continue.
        // If the ray does not hit another occluder, the light's attenuation from this glass shell
        // (along with other glass shells) is then used.
        optixIgnoreIntersection();
}

extern "C" __global__ void __miss__constant_bg()
{
    whitted::PayloadRadiance prd = getPayloadRadiance();
    prd.result                   = params.miss_color;
    setPayloadRadiance( prd );
}
