#include "hip/hip_runtime.h"
#pragma once

#include <cuda/helpers.h>
#include <hip/hip_runtime.h>
#include <hip/hip_vector_types.h>

#include <cuda/std/complex>

using cuda::std::complex;

// struct complexFloat {
//	float real;
//	float img;
// };

struct complexFloat3 {
    complex<float> x, y, z;
};

//__device__ complexFloat makeComplexFloat(float r, float i) {
//	complexFloat res;
//	res.real = r;
//	res.img = i;
//	return res;
//}
//
__device__ complexFloat3 makeComplexFloat3(complex<float> a, complex<float> b,
                                           complex<float> c) {
    complexFloat3 res;
    res.x = a;
    res.y = b;
    res.z = c;
    return res;
}

__device__ __forceinline void printComplexFloat3(complexFloat3 cf3) {
    printf("((%7f,%7f),(%7f,%7f),(%7f,%7f))\n", cf3.x.real(), cf3.x.imag(),
           cf3.y.real(), cf3.y.imag(), cf3.z.real(), cf3.z.imag());
}

//
//__device__ complexFloat exp(complexFloat cf) {
//	float a = cf.real;
//	float b = cf.img;
//	float aexp = exp(a);
//	float real = aexp * cosf(b);
//	float img = aexp * sinf(b);
//	return makeComplexFloat(real, img);
//}
//
//
//__device__ complexFloat operator-(complexFloat cf) {
//	return makeComplexFloat(-cf.real, -cf.img);
//}

__device__ complexFloat3 operator-(complexFloat3 cf3) {
    return makeComplexFloat3(-cf3.x, -cf3.y, -cf3.z);
}

//__device__ complexFloat operator*(const complexFloat& a, const float& b)
//{
//	return makeComplexFloat(a.real * b, a.img * b);
//}
//
//
__device__ complexFloat3 operator*(complex<float> cf, float3 f3) {
    complex<float> a = cf * f3.x;
    complex<float> b = cf * f3.y;
    complex<float> c = cf * f3.z;
    return makeComplexFloat3(a, b, c);
}

__device__ complexFloat3 operator-(complexFloat3 a, complexFloat3 b) {
    complex<float> cfa = a.x - b.x;
    complex<float> cfb = a.y - b.y;
    complex<float> cfc = a.z - b.z;
    return makeComplexFloat3(cfa, cfb, cfc);
}

__device__ complexFloat3 operator*(complexFloat3 cf3, float f) {
    complex<float> a = cf3.x * f;
    complex<float> b = cf3.y * f;
    complex<float> c = cf3.z * f;
    return makeComplexFloat3(a, b, c);
}

__device__ complexFloat3 operator+(complexFloat3 cf3a, complexFloat3 cf3b) {
    complex<float> a = cf3a.x + cf3b.x;
    complex<float> b = cf3a.y + cf3b.y;
    complex<float> c = cf3a.z + cf3b.z;
    return makeComplexFloat3(a, b, c);
}

__device__ complexFloat3 cross(complexFloat3 a, float3 b) {
    // return make_float3(a.y * b.z - a.z * b.y, a.z * b.x - a.x * b.z, a.x *
    // b.y - a.y * b.x);
    complex<float> cfa = a.y * b.z - a.z * b.y;
    complex<float> cfb = a.z * b.x - a.x * b.z;
    complex<float> cfc = a.x * b.y - a.y * b.x;
    return makeComplexFloat3(cfa, cfb, cfc);
}
//
//
__device__ complex<float> dot(complexFloat3 cf3, float3 f3) {
    return cf3.x * f3.x + cf3.y * f3.y + cf3.z * f3.z;
}

//// CtsToSph
//// Physics convention
//// X Y Z --> R Phi(x-y angle, 0 to 2pi) Theta(z-xy angle, 0 to pi)
__device__ float3 CtsToSph(float3 f3) {
    float x = f3.x;
    float y = f3.y;
    float z = f3.z;
    float r = sqrt(x * x + y * y + z * z);
    float theta = acosf(z / r);
    float phi = atan2f(y, x);

    return make_float3(r, phi, theta);
}

//__device__  complex<float> multiply(complex<float> cf1, complexFloat cf2) {
//	// assume (a+bi)*(c+di)
//    // ac - bd
//	float a = cf1.real;
//	float b = cf1.img;
//	float c = cf2.real;
//	float d = cf2.img;
//	float real = a * c - b * d;
//	// bc + ad
//	float img = b * c + a * d;
//	return makeComplexFloat(real, img);
//
//}

inline __device__ void OrthonormalSet(float angP, float angT, float3& dirN,
                                      float3& dirU, float3& dirR) {
    printf("OrthonormalSet start\n");
    float cp = sinf(angP);
    float sp = sinf(angP);
    float ct = cosf(angT);
    float st = sinf(angT);

    dirN.x = st * cp;
    dirN.y = st * sp;
    dirN.z = ct;

    dirR.x = sp;
    dirR.y = -cp;
    dirR.z = 0;

    dirU = cross(dirR, dirN);
}

// OrthonormalSet N=2
inline __device__ void OrthonormalSet(float ang, float2 dirN, float2 dirR) {
    float c = cosf(ang);
    float s = sinf(ang);

    dirN.x = c;
    dirN.y = s;

    dirR.x = s;
    dirR.y = -c;
}

inline __device__ void Orthonormalize(float3& dirN, float3& dirU,
                                      float3& dirR) {
    dirN = normalize(dirN);
    dirU = normalize(dirU);
    dirR = normalize(dirR);

    dirU = normalize(dirU - dot(dirU, dirN) * dirN);

    dirR = normalize(dirR - dot(dirR, dirN) * dirN);
    dirR = normalize(dirR - dot(dirR, dirU) * dirU);
}